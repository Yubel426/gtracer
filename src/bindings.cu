#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>

#include <cstdint>
#include <cmath>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <glm/glm.hpp>

#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <gtracer/bvh.h>

namespace py = pybind11;
namespace gtracer {

class GaussianTracer {
public:
    GaussianTracer(){
        triangle_bvh = TriangleBvhBase::make();
    }

    void build_bvh(const torch::Tensor& triangles){
        const size_t n_triangles = triangles.size(0);
        hipStream_t m_stream = at::cuda::getCurrentCUDAStream();;
        triangle_bvh->build_bvh(triangles.data_ptr<float>(), n_triangles, m_stream);
    }

    void update_bvh(const torch::Tensor& triangles){
        const size_t n_triangles = triangles.size(0);
        hipStream_t m_stream = at::cuda::getCurrentCUDAStream();;
        triangle_bvh->update_bvh(triangles.data_ptr<float>(), n_triangles, m_stream);
    }

    void trace_forward(
        const torch::Tensor rays_o, const torch::Tensor rays_d, const torch::Tensor gs_idxs, 
        const torch::Tensor faces, const torch::Tensor vertices, const torch::Tensor opacity, const torch::Tensor SinvR, const torch::Tensor shs, 
        torch::Tensor colors, torch::Tensor depth, torch::Tensor alpha, 
        const float alpha_min, const float transmittance_min, const int deg
        ){
        const uint32_t n_elements = rays_o.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        int max_coeffs = shs.size(1);

        triangle_bvh->gaussian_trace_forward(
            n_elements, (const glm::vec3*)rays_o.data_ptr<float>(), (const glm::vec3*)rays_d.data_ptr<float>(), gs_idxs.data_ptr<int>(), 
            (const int3*)faces.data_ptr<int>(), (const glm::vec3*)vertices.data_ptr<float>(), opacity.data_ptr<float>(), (const glm::mat3x3*)SinvR.data_ptr<float>(), (const glm::vec3*)shs.data_ptr<float>(), 
            (glm::vec3*)colors.data_ptr<float>(), depth.data_ptr<float>(), alpha.data_ptr<float>(), 
            alpha_min, transmittance_min, deg, max_coeffs, stream);
    }
    
    void trace_backward(
        const torch::Tensor rays_o, const torch::Tensor rays_d, const torch::Tensor gs_idxs, 
        const torch::Tensor means3D, const torch::Tensor opacity, const torch::Tensor SinvR, const torch::Tensor shs, 
        const torch::Tensor colors, const torch::Tensor depth, const torch::Tensor alpha, 
        torch::Tensor grad_means3D, torch::Tensor grad_opacity, torch::Tensor grad_SinvR, torch::Tensor grad_shs, 
        const torch::Tensor grad_out_color, const torch::Tensor grad_out_depth, const torch::Tensor grad_out_alpha,
        const float alpha_min, const float transmittance_min, const int deg
        ){
        const uint32_t n_elements = rays_o.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        int max_coeffs = shs.size(1);

        triangle_bvh->gaussian_trace_backward(
            n_elements, (const glm::vec3*)rays_o.data_ptr<float>(), (const glm::vec3*)rays_d.data_ptr<float>(), gs_idxs.data_ptr<int>(), 
            (const glm::vec3*)means3D.data_ptr<float>(), opacity.data_ptr<float>(), (const glm::mat3x3*)SinvR.data_ptr<float>(), (const glm::vec3*)shs.data_ptr<float>(), 
            (const glm::vec3*)colors.data_ptr<float>(), depth.data_ptr<float>(), alpha.data_ptr<float>(), 
            (glm::vec3*)grad_means3D.data_ptr<float>(), grad_opacity.data_ptr<float>(), (glm::mat3x3*)grad_SinvR.data_ptr<float>(), (glm::vec3*)grad_shs.data_ptr<float>(), 
            (const glm::vec3*)grad_out_color.data_ptr<float>(), grad_out_depth.data_ptr<float>(), grad_out_alpha.data_ptr<float>(),
            alpha_min, transmittance_min, deg, max_coeffs, stream);
    }

    std::shared_ptr<TriangleBvhBase> triangle_bvh;
};

GaussianTracer* create_gaussiantracer() {
    return new GaussianTracer{};
}

}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {

py::class_<gtracer::GaussianTracer>(m, "GaussianTracer")
    .def("trace_forward", &gtracer::GaussianTracer::trace_forward)
    .def("trace_backward", &gtracer::GaussianTracer::trace_backward)
    .def("build_bvh", &gtracer::GaussianTracer::build_bvh)
    .def("update_bvh", &gtracer::GaussianTracer::update_bvh);

m.def("create_gaussiantracer", &gtracer::create_gaussiantracer);

}